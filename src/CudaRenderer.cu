#include "hip/hip_runtime.h"
#include "CudaRenderer.hpp"

#include <GL/glew.h>
#include <GL/gl.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include "Utils.hpp"
#include "Triangle.hpp"

#define BLOCKWIDTH 8
#define INTERSECT_EPSILON 0.0000001f
#define OFFSET_EPSILON 0.00001f
#define BIGT 99999.f
#define AIR_INDEX 1.f

template<class T> __device__ inline T operator~ (T a) { return (T)~(int)a; }
template<class T> __device__ inline T operator| (T a, T b) { return (T)((int)a | (int)b); }
template<class T> __device__ inline T operator& (T a, T b) { return (T)((int)a & (int)b); }
template<class T> __device__ inline T operator^ (T a, T b) { return (T)((int)a ^ (int)b); }
template<class T> __device__ inline T& operator|= (T& a, T b) { return (T&)((int&)a |= (int)b); }
template<class T> __device__ inline T& operator&= (T& a, T b) { return (T&)((int&)a &= (int)b); }
template<class T> __device__ inline T& operator^= (T& a, T b) { return (T&)((int&)a ^= (int)b); }

#define LEFT_HIT_BIT 0x80000000
#define RIGHT_HIT_BIT 0x40000000

__device__ inline float scramble(const uint32_t scrambleConstant, const float f)
{
	const uint32_t i = static_cast<uint32_t>(f * 0x100000000) ^ scrambleConstant;

	const float r = i * 2.3283064365386963e-10f;

	return r;
}

__device__ bool bboxIntersect(const AABB box, const float3 origin,
		const float3 inverseDirection, float& t)
{
	float3 tmin = make_float3(-BIGT, -BIGT, -BIGT), tmax = make_float3(BIGT,
	BIGT,
	BIGT);

	const float3 tdmin = (box.min - origin) * inverseDirection;
	const float3 tdmax = (box.max - origin) * inverseDirection;

	tmin = fminf(tdmin, tdmax);
	tmax = fmaxf(tdmin, tdmax);

	const float tmind = fmin_compf(tmin);
	const float tmaxd = fmin_compf(tmax);

	t = fminf(tmind, tmaxd);

	return tmaxd >= tmind && !(tmaxd < 0.f && tmind < 0.f);
}

__device__ bool rayTriangleIntersection(const Ray ray, const Triangle& triangle,
		float& t, float2& uv)
{
	/* Möller-Trumbore algorithm
	 * https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
	 */

	// TODO: Experiment with __ldg
	const float3 vertex0 = triangle.vertices[0].p;

	const float3 edge1 = triangle.vertices[1].p - vertex0;
	const float3 edge2 = triangle.vertices[2].p - vertex0;

	const float3 h = cross(ray.direction, edge2);
	const float a = dot(edge1, h);

	if (a > -INTERSECT_EPSILON && a < INTERSECT_EPSILON)
		return false;

	const float f = __fdividef(1.f, a);
	const float3 s = ray.origin - vertex0;
	const float u = f * dot(s, h);

	if (u < 0.f || u > 1.0f)
		return false;

	const float3 q = cross(s, edge1);
	const float v = f * dot(ray.direction, q);

	if (v < 0.0 || u + v > 1.0)
		return false;

	t = f * dot(edge2, q);

	if (t > INTERSECT_EPSILON)
	{
		uv = make_float2(u, v);
		return true;
	}
	else
		return false;
}

enum HitType
{
	ANY, CLOSEST
};

template<const HitType hitType>
__device__ RaycastResult rayCast(const Ray ray, const Node* bvh,
		const Triangle* triangles, const float maxT)
{
	float tMin = maxT;
	int32_t minTriIdx = -1;
	float2 minUV;
	RaycastResult result;
	const float3 inverseDirection = make_float3(1.f, 1.f, 1.f) / ray.direction;

	int32_t ptr = 0;
	unsigned int stack[16] { 0 };
	int32_t i = -1;
	float t = 0;
	float2 uv;
	bool getNextNode = true;

	while (ptr >= 0)
	{
		uint32_t currentNodeIdx = stack[ptr];
		Node currentNode = bvh[currentNodeIdx];

		if (currentNode.rightIndex == -1)
		{
			getNextNode = false;

			if (i >= currentNode.startTri
					&& i < currentNode.startTri + currentNode.nTri)
			{
				if (rayTriangleIntersection(ray, triangles[i], t, uv))
				{

					if (t < tMin)
					{
						tMin = t;
						minTriIdx = i;
						minUV = uv;

						if (hitType == HitType::ANY)
							break;
					}
				}

				++i;

				if (i >= currentNode.startTri + currentNode.nTri)
					getNextNode = true;

			}
			else
			{
				i = currentNode.startTri;
			}

		}
		else
		{
			const AABB leftBox = bvh[stack[ptr] + 1].bbox;
			const AABB rightBox = bvh[currentNode.rightIndex].bbox;

			float leftt, rightt;

			uint32_t hitMask =
					bboxIntersect(leftBox, ray.origin, inverseDirection,
							leftt) ?
					LEFT_HIT_BIT :
										0x00;
			hitMask =
					bboxIntersect(rightBox, ray.origin, inverseDirection,
							rightt) ? hitMask | RIGHT_HIT_BIT : hitMask;

			// TODO: Push closer one last, don't intersect if closest hit is closer than box
			if ((hitMask & LEFT_HIT_BIT) != 0x00 && leftt < tMin)
			{
				stack[ptr] = currentNodeIdx + 1;
				++ptr;
			}

			if ((hitMask & RIGHT_HIT_BIT) != 0x00 && rightt < tMin)
			{
				stack[ptr] = currentNode.rightIndex;
				++ptr;
			}
		}

		if (getNextNode)
		{
			--ptr;
			i = -1;
		}

	}

	if (minTriIdx == -1)
		return result;

	result.t = tMin;
	result.triangleIdx = minTriIdx;
	result.uv = minUV;

	return result;
}

__device__ void writeToCanvas(const uint32_t x, const uint32_t y,
		const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize,
		const float3 data)
{
	const float4 out = make_float4(data.x, data.y, data.z, 1.f);
	surf2Dwrite(out, surfaceObj, (canvasSize.x - 1 - x) * sizeof(out), y);
	return;
}

__device__ float3 readFromCanvas(const uint32_t x, const uint32_t y,
		const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize)
{
	float4 in;
	surf2Dread(&in, surfaceObj, (canvasSize.x - 1 - x) * sizeof(in), y);
	const float3 ret = make_float3(in.x, in.y, in.z);

	return ret;
}

__global__ void logicKernel(const glm::ivec2 canvasSize, Queues queues,
		Paths paths, const Material* materials,
		const uint32_t* triangleMaterialIds)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t idx = x + y * canvasSize.x;

	const float3 float3_zero = make_float3(0.f, 0.f, 0.f);

	if (x >= canvasSize.x || y >= canvasSize.y)
		return;

	const float rf = paths.floats[0];

	const RaycastResult result = paths.result[idx];
	const uint32_t rayNr = paths.rayNr[idx];

	if (!result || rayNr >= 5)
	{
		const uint32_t new_idx = atomicAdd(queues.newPathQueueSize, 1);
		queues.newPathQueue[new_idx] = idx;
		paths.color[idx] = make_float3(0.2f, 0.2f, 0.2f);
		return;
	}

	const Material material = materials[triangleMaterialIds[result.triangleIdx]];

	uint32_t new_idx;

	switch (material.mode)
	{
		case (Material::TRANSPARENCY_REFLECTION_FRESNEL):
			{
				const float total = length(material.colorDiffuse) + length(material.colorSpecular) + length(material.colorTransparent);
				const float diffuseTreshold = length(material.colorDiffuse) / total;
				const float specularTreshold = diffuseTreshold + length(material.colorSpecular) / total;
				const float transparentTreshold = diffuseTreshold + specularTreshold + length(material.colorTransparent) / total;

				if (rf < diffuseTreshold)
				{
					new_idx = atomicAdd(queues.diffuseQueueSize, 1);
					queues.diffuseQueue[new_idx] = idx;
				}else if (rf > diffuseTreshold && rf < specularTreshold)
				{
					 new_idx = atomicAdd(queues.specularQueueSize, 1);
					 queues.specularQueue[new_idx] = idx;
				}else
				{
					 new_idx = atomicAdd(queues.transparentQueueSize, 1);
					 queues.transparentQueue[new_idx] = idx;
				}
			}
			break;

		case (Material::REFLECTION_FRESNEL):
			{
				const float total = length(material.colorDiffuse) + length(material.colorSpecular);
				const float diffuseTreshold = length(material.colorDiffuse) / total;
				const float specularTreshold = diffuseTreshold + length(material.colorSpecular) / total;

				if (rf < diffuseTreshold)
				{
					new_idx = atomicAdd(queues.diffuseQueueSize, 1);
					queues.diffuseQueue[new_idx] = idx;
				}else
				{
					 new_idx = atomicAdd(queues.specularQueueSize, 1);
					 queues.specularQueue[new_idx] = idx;
				}
			}
			break;

		default:
			new_idx = atomicAdd(queues.diffuseQueueSize, 1);
			queues.diffuseQueue[new_idx] = idx;
			break;
	}

	return;
}

__global__ void writeToCanvas(const glm::ivec2 canvasSize,
		hipSurfaceObject_t canvas, Paths paths)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const int idx = x + y * canvasSize.x;

	if (x >= canvasSize.x || y >= canvasSize.y)
		return;

	const uint32_t currentPath = paths.pathNr[idx];
	const float3 newColor = paths.color[idx];
	float3 oldColor = readFromCanvas(x, y, canvas, canvasSize);
	float3 blend = static_cast<float>(currentPath - 1) / currentPath * oldColor
			+ 1.f / currentPath * newColor;

	writeToCanvas(x, y, canvas, canvasSize, blend);
}

typedef struct
{
	float3 col[3];
} float33;

__device__
inline void setZero(float33& m)
{
	m.col[0] = make_float3(0.0f, 0.0f, 0.0f);
	m.col[1] = make_float3(0.0f, 0.0f, 0.0f);
	m.col[2] = make_float3(0.0f, 0.0f, 0.0f);
}

__device__
  inline float3 operator*(const float33 m, const float3 v)
{
	float3 res;
	res.x = m.col[0].x * v.x + m.col[1].x * v.y + m.col[2].x * v.z;
	res.y = m.col[0].y * v.x + m.col[1].y * v.y + m.col[2].y * v.z;
	res.z = m.col[0].z * v.x + m.col[1].z * v.y + m.col[2].z * v.z;

	return res;
}

__device__ float33 getBasis(const float3 n)
{

	float33 R;

	float3 Q = n;
	const float3 absq = abs(Q);
	float absqmin = fmin(absq);

	if (absq.x == absqmin)
		Q.x = 1;
	else if (absq.y == absqmin)
		Q.y = 1;
	else
		Q.z = 1;

	float3 T = normalize(cross(Q, n));
	float3 B = normalize(cross(n, T));

	R.col[0] = T;
	R.col[1] = B;
	R.col[2] = n;

	return R;
}

__global__ void diffuseKernel(const glm::ivec2 canvasSize, const Queues queues,
		Paths paths, const Triangle* triangles,
		const uint32_t* lightTriangleIds, const uint32_t lightTriangles,
		const uint32_t* triangleMaterialIds, const Material* materials,
		const Node* bvh)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t idx = x + y * canvasSize.x;

	if (idx >= *queues.diffuseQueueSize)
		return;

	const uint32_t pathIdx = queues.diffuseQueue[idx];
	const uint32_t scrambleConstant = paths.scrambleConstants[pathIdx];

	const RaycastResult result = paths.result[pathIdx];
	const Material material = materials[triangleMaterialIds[result.triangleIdx]];
	const Ray ray = paths.ray[pathIdx];
	const float3 hitNormal = triangles[result.triangleIdx].normal();

	const float3 shadowRayOrigin = ray.origin + ray.direction*result.t + hitNormal * OFFSET_EPSILON;

	float3 brightness = make_float3(0.f, 0.f, 0.f);

	for (uint32_t i = 0; i < lightTriangles; ++i)
	{
		float pdf;
		float3 shadowPoint;

		float r0 = paths.floats[1+2*i];
		float r1 = paths.floats[2+2*i];

		r0 = scramble(scrambleConstant, r0);
		r1 = scramble(scrambleConstant, r1);

		triangles[lightTriangleIds[i]].sample(pdf, shadowPoint, r0, r1);

		const float3 shadowRayDirection = shadowPoint - shadowRayOrigin;
		const Ray shadowRay(shadowRayOrigin, normalize(shadowRayDirection));
		const float shadowRayLength = length(shadowRayDirection);

		const Triangle lightTriangle = triangles[lightTriangleIds[i]];
		const Material lightTriangleMaterial = materials[triangleMaterialIds[lightTriangleIds[i]]];
		const float3 lightEmission = lightTriangleMaterial.colorEmission;

		RaycastResult shadowResult = rayCast<HitType::ANY>(shadowRay, bvh, triangles, shadowRayLength);

		if ((shadowResult && shadowResult.t >= shadowRayLength + OFFSET_EPSILON) || !shadowResult)
		{
			const float cosOmega = __saturatef(dot(normalize(shadowRayDirection), hitNormal));
			const float cosL = __saturatef(dot(-normalize(shadowRayDirection), lightTriangle.normal()));

			brightness += 1.f / (shadowRayLength * shadowRayLength * pdf) * lightEmission * cosL * cosOmega;
		}
	}

	const float3 currentTroughput = paths.throughput[pathIdx];
	const float3 filteredAmbient = currentTroughput * material.colorAmbient;
	const float3 filteredDiffuse = currentTroughput * material.colorDiffuse;
	const float3 fiteredEmission = currentTroughput * material.colorEmission; // TODO: Don't add this, light is sampled explicitly

	paths.color[pathIdx] += fiteredEmission + filteredAmbient + brightness / lightTriangles * filteredDiffuse / HIP_PI_F;
}

__global__ void newPathsKernel(const glm::ivec2 canvasSize, const Queues queues,
		const Paths paths, const Camera camera)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const int idx = x + y * canvasSize.x;

	if (idx >= *queues.newPathQueueSize)
		return;

	const uint32_t pathIdx = queues.newPathQueue[idx];

	const uint32_t xCoordinate = pathIdx % canvasSize.x;
	const uint32_t yCoordinate = pathIdx / canvasSize.x;

	const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(
			xCoordinate, yCoordinate, canvasSize);
	const Ray ray = camera.generateRay(nic, static_cast<float>(canvasSize.x) / canvasSize.y);

	paths.ray[pathIdx] = ray;

	paths.color[pathIdx] = make_float3(0.f, 0.f, 0.f);
	paths.throughput[pathIdx] = make_float3(1.f, 1.f, 1.f);
	paths.p[pathIdx] = 1.f;
	paths.rayNr[pathIdx] = 1;
	paths.pathNr[pathIdx] += 1;
}

inline __device__ float3 reflectionDirection(const float3 normal,
		const float3 incomingDirection)
{

	const float cosT = dot(incomingDirection, normal);

	return incomingDirection - 2 * cosT * normal;
}

inline __device__ float3 refractionDirection(const float cosInAng, const float sin2t, const float3 normal, const float3 incoming, const float index1, const float index2)
{
    return index1 / index2 * incoming + (index1 / index2 * cosInAng - sqrt(1 - sin2t)) * normal;
}

__global__ void diffuseExtensionKernel(const glm::ivec2 canvasSize,
		const Queues queues, const Paths paths, const Triangle* triangles,
		const uint32_t* triangleMaterialIds, const Material* materials, const uint32_t lightTriangles)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t idx = x + y * canvasSize.x;

	if (idx >= *queues.diffuseQueueSize)
		return;

	const uint32_t pathIdx = queues.diffuseQueue[idx];
	const uint32_t scrambleConstant = paths.scrambleConstants[idx];

	const Ray ray = paths.ray[pathIdx];
	const RaycastResult result = paths.result[pathIdx];
	const Triangle triangle = triangles[result.triangleIdx];
	const Material& material = materials[triangleMaterialIds[result.triangleIdx]];
	float3 hitNormal = triangle.normal();

	float33 B = getBasis(hitNormal);

	float r0 = paths.floats[1+2*lightTriangles];
	float r1 = paths.floats[2+2*lightTriangles];

	r0 = scramble(scrambleConstant, r0);
	r1 = scramble(scrambleConstant, r1);

	const float sinTheta = sqrtf(r0);
	const float cosTheta = sqrtf(1-sinTheta*sinTheta);

	const float psi = r1*2*HIP_PI_F;

	float3 extensionDir = make_float3(sinTheta*cosf(psi), sinTheta*sinf(psi), cosTheta);

	extensionDir = B * extensionDir;
	extensionDir = normalize(extensionDir); // Unnecessary
	const float3 extensionOrig = ray.origin + ray.direction*result.t + OFFSET_EPSILON * hitNormal;
	const Ray extensionRay(extensionOrig, extensionDir);

	float cosO = dot(extensionDir, hitNormal);
	float p = cosO * dot(extensionDir, hitNormal) * (1.f / HIP_PI_F);
	float3 throughput = material.colorDiffuse / HIP_PI_F * dot(extensionDir, hitNormal);

	paths.ray[pathIdx] = extensionRay;
	paths.throughput[pathIdx] = paths.throughput[pathIdx] * throughput;
	paths.p[pathIdx] *= p;
	paths.rayNr[pathIdx] += 1;
}

__global__ void
 specularKernel(
	 const glm::ivec2 canvasSize,
	 const Queues queues,
	 const Paths paths,
	 const Triangle* triangles,
	 const uint32_t* triangleMaterialIds,
	 const Material* materials
	 )
 {
	 const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	 const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	 const uint32_t idx = x + y * canvasSize.x;

	 if (idx >= *queues.specularQueueSize)
		 return;

	 const float3 float3_zero = make_float3(0.f, 0.f, 0.f);
	 const uint32_t pathIdx = queues.specularQueue[idx];

	 const Ray hitRay = paths.ray[pathIdx];
	 const RaycastResult hitResult = paths.result[pathIdx];
	 const float3 materialSpecularColor = materials[triangleMaterialIds[hitResult.triangleIdx]].colorSpecular;

	 const Triangle triangle = triangles[hitResult.triangleIdx];
	 const float3 hitNormal = triangle.normal(hitResult.uv);

	 const float3 reflectionRayOrigin = hitRay.origin + hitRay.direction*hitResult.t + hitNormal * OFFSET_EPSILON;
	 const float3 reflectionRayDir = reflectionDirection(hitNormal, hitRay.direction);

	 const Ray reflectionRay(reflectionRayOrigin, reflectionRayDir);

	 paths.ray[pathIdx] = reflectionRay;
	 paths.throughput[pathIdx] = paths.throughput[pathIdx] * materialSpecularColor;
	 paths.rayNr[pathIdx] += 1;
 }

__global__ void
 transparentKernel(
	 const glm::ivec2 canvasSize,
	 const Queues queues,
	 const Paths paths,
	 const Triangle* triangles,
	 const uint32_t* triangleMaterialIds,
	 const Material* materials
	 )
 {
	 const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	 const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	 const uint32_t idx = x + y * canvasSize.x;

	 if (idx >= *queues.transparentQueueSize)
		 return;

	 const float3 float3_zero = make_float3(0.f, 0.f, 0.f);
	 const uint32_t pathIdx = queues.transparentQueue[idx];

	 const Ray hitRay = paths.ray[pathIdx];
	 const RaycastResult hitResult = paths.result[pathIdx];

	 const Triangle triangle = triangles[hitResult.triangleIdx];
	 float3 hitNormal = triangle.normal(hitResult.uv);
	 const Material material = materials[triangleMaterialIds[hitResult.triangleIdx]];

     const float idx1 = AIR_INDEX;
     const float idx2 = material.refractionIndex;

     float rat;

     const bool outside = dot(hitRay.direction, hitNormal) < 0.f;

     if (outside)
       rat = idx2 / idx1;
     else
     {
       rat = idx1 / idx2;
       hitNormal = -hitNormal;
     }

     const float cosi = fabsf(dot(hitRay.direction, hitNormal));

     if (sinf(acosf(cosi)) <= rat) // Check for total internal reflection
     {
		const float sin2t = fabs((idx1 / idx2) * (idx1 / idx2) * (1 - cosi * cosi));

		const float3 transOrig = hitRay.origin + hitRay.direction*hitResult.t - hitNormal * OFFSET_EPSILON;
		const float3 transDir = refractionDirection(cosi, sin2t, hitNormal, hitRay.direction, idx1, idx2);

		paths.ray[pathIdx] = Ray(transOrig, transDir);
		paths.throughput[pathIdx] = paths.throughput[pathIdx] * material.colorTransparent;
		paths.rayNr[pathIdx] += 1;
     }else
     {
		const float3 reflectionRayOrigin = hitRay.origin + hitRay.direction*hitResult.t + hitNormal * OFFSET_EPSILON;
		const float3 reflectionRayDir = reflectionDirection(hitNormal, hitRay.direction);

		const Ray reflectionRay(reflectionRayOrigin, reflectionRayDir);

		paths.ray[pathIdx] = reflectionRay;
		paths.throughput[pathIdx] = paths.throughput[pathIdx] * material.colorSpecular;
		paths.rayNr[pathIdx] += 1;
     }
 }

__global__ void resetAllPaths(Paths paths, Camera camera,
		const glm::fvec2 canvasSize)
{
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const int idx = x + y * canvasSize.x;

	if (x >= canvasSize.x || y >= canvasSize.y)
		return;

	const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(
			x, y, canvasSize);
	const Ray ray = camera.generateRay(nic,
			static_cast<float>(canvasSize.x) / canvasSize.y);

	paths.ray[idx] = ray;
	paths.color[idx] = make_float3(0.f, 0.f, 0.f);
	paths.throughput[idx] = make_float3(1.f, 1.f, 1.f);
	paths.p[idx] = 1.f;
	paths.rayNr[idx] = 1;
	paths.pathNr[idx] = 1;
}

__global__ void castRays(Paths paths, const glm::ivec2 canvasSize,
		const Triangle* triangles, const Node* bvh, const Material* materials,
		const unsigned int* traingelMaterialIds)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int idx = x + y * canvasSize.x;

	if (x >= canvasSize.x || y >= canvasSize.y)
		return;

	const Ray ray = paths.ray[idx];
	RaycastResult result = rayCast<HitType::CLOSEST>(ray, bvh, triangles, BIGT);
	paths.result[idx] = result;
}

__global__ void generateRandom(hiprandStateSobol32* state, const glm::ivec2 size,
		const uint32_t nRands, float* output)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= size.x || y >= size.y)
		return;

	hiprandStateSobol32 localState = state[x + size.x * y];

	for (uint32_t i = 0; i < nRands; ++i)
		output[nRands * (x + y * size.x) + i] = hiprand_uniform(&localState);

	state[x + size.x * y] = localState;
}

__global__ void initRand(const int seq, hiprandState_t* const curandStateDevPtr,
		const glm::ivec2 size)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= size.x || y >= size.y)
		return;

	hiprandState_t localState;
	hiprand_init(x + y * size.x, seq, 0, &localState);
	curandStateDevPtr[x + y * size.x] = localState;
}

void CudaRenderer::reset()
{
	queues.reset();
	callcntr = 0;

	dim3 block(BLOCKWIDTH, BLOCKWIDTH);
	dim3 grid((lastSize.x + block.x - 1) / block.x,
			(lastSize.y + block.y - 1) / block.y);

	resetAllPaths<<<grid, block>>>(paths, lastCamera, lastSize);
	CUDA_CHECK(hipDeviceSynchronize());
}

void CudaRenderer::resize(const glm::ivec2 size)
{
	queues.resize(size);
	paths.resize(size);

	lastSize = size;

	dim3 block(BLOCKWIDTH, BLOCKWIDTH);
	dim3 grid((size.x + block.x - 1) / block.x,
			(size.y + block.y - 1) / block.y);

	uint32_t* hostScrambleConstants;

	CURAND_CHECK(hiprandGetScrambleConstants32(&hostScrambleConstants));
	CUDA_CHECK(hipMemcpy(paths.scrambleConstants, hostScrambleConstants, size.x * size.y * sizeof(uint32_t), hipMemcpyHostToDevice));

	reset();
}

CudaRenderer::CudaRenderer() :
		lastCamera(), lastSize(), callcntr(0)
{
	uint32_t cudaDeviceCount = 0;
	int cudaDevices[8];
	uint32_t cudaDevicesCount = 8;

	hipGLGetDevices(&cudaDeviceCount, cudaDevices, cudaDevicesCount,
			hipGLDeviceListCurrentFrame);

	if (cudaDeviceCount < 1)
		throw std::runtime_error("No CUDA devices available");


	CURAND_CHECK(hiprandCreateGenerator(&randGen, HIPRAND_RNG_QUASI_SOBOL32));
	CURAND_CHECK(hiprandSetQuasiRandomGeneratorDimensions(randGen, 32));

	CUDA_CHECK(hipSetDevice(cudaDevices[0]));

	resize(glm::ivec2(WWIDTH, WHEIGHT));
}

CudaRenderer::~CudaRenderer()
{
	queues.release();
	paths.release();
}

void CudaRenderer::pathTraceToCanvas(GLTexture& canvas, const Camera& camera,
		Model& model)
{
	if (model.getNTriangles() == 0)
		return;

	const glm::ivec2 canvasSize = canvas.getSize();
	const bool diffCamera = std::memcmp(&camera, &lastCamera, sizeof(Camera));
	const bool diffSize = (canvasSize != lastSize);
	const auto surfaceObj = canvas.getCudaMappedSurfaceObject();

	const dim3 block(BLOCKWIDTH, BLOCKWIDTH);
	const dim3 grid((canvasSize.x + block.x - 1) / block.x,
			(canvasSize.y + block.y - 1) / block.y);

	if (diffCamera != 0 || diffSize != 0)
	{
		lastCamera = camera;

		reset();
	}

    CURAND_CHECK(hiprandGenerateUniform(randGen, paths.floats, 32));

	castRays<<<grid, block>>>(paths, canvasSize, model.getDeviceTriangles(),
			model.getDeviceBVH(), model.getDeviceMaterials(),
			model.getDeviceTriangleMaterialIds());

	CUDA_CHECK(hipDeviceSynchronize());

	logicKernel<<<grid, block>>>(canvasSize, queues, paths,
			model.getDeviceMaterials(), model.getDeviceTriangleMaterialIds());

	CUDA_CHECK(hipDeviceSynchronize());

	diffuseKernel<<<grid, block>>>(canvasSize, queues, paths,
			model.getDeviceTriangles(), model.getDeviceLightIds(),
			model.getNLights(), model.getDeviceTriangleMaterialIds(),
			model.getDeviceMaterials(), model.getDeviceBVH());

	CUDA_CHECK(hipDeviceSynchronize());

	specularKernel<<<grid, block>>>(
		canvasSize,
		queues,
		paths,
		model.getDeviceTriangles(),
		model.getDeviceTriangleMaterialIds(),
		model.getDeviceMaterials()
	);

	 CUDA_CHECK(hipDeviceSynchronize());
	 CUDA_CHECK(hipMemset(queues.specularQueueSize, 0, sizeof(uint32_t)));

	transparentKernel<<<grid, block>>>(
		canvasSize,
		queues,
		paths,
		model.getDeviceTriangles(),
		model.getDeviceTriangleMaterialIds(),
		model.getDeviceMaterials()
	 );

	 CUDA_CHECK(hipDeviceSynchronize());
	 CUDA_CHECK(hipMemset(queues.transparentQueueSize, 0, sizeof(uint32_t)));

	writeToCanvas<<<grid, block>>>(canvasSize, surfaceObj, paths);

	diffuseExtensionKernel<<<grid, block>>>(canvasSize, queues, paths,
			model.getDeviceTriangles(), model.getDeviceTriangleMaterialIds(),
			model.getDeviceMaterials(), model.getNLights());

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemset(queues.diffuseQueueSize, 0, sizeof(uint32_t)));

	newPathsKernel<<<grid, block>>>(canvasSize, queues, paths, camera);

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemset(queues.newPathQueueSize, 0, sizeof(uint32_t)));

	canvas.cudaUnmap();
}

