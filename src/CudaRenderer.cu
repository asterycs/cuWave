#include "hip/hip_runtime.h"
#include "CudaRenderer.hpp"

#include <GL/glew.h>
#include <GL/gl.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include "Utils.hpp"
#include "Triangle.hpp"

#define BLOCKWIDTH 8
#define INTERSECT_EPSILON 0.0000001f
#define OFFSET_EPSILON 0.00001f
#define BIGT 99999.f
#define AIR_INDEX 1.f

#define REFLECTIVE_BIT 0x80000000
#define REFRACTIVE_BIT 0x40000000
#define INSIDE_BIT 0x20000000

#define LEFT_HIT_BIT 0x80000000
#define RIGHT_HIT_BIT 0x40000000

__device__ uint32_t mix(uint32_t a, uint32_t b, uint32_t c)
{
  a -= b; a -= c; a ^= (c>>13);
  b -= c; b -= a; b ^= (a<<8);
  c -= a; c -= b; c ^= (b>>13);
  a -= b; a -= c; a ^= (c>>12);
  b -= c; b -= a; b ^= (a<<16);
  c -= a; c -= b; c ^= (b>>5);
  a -= b; a -= c; a ^= (c>>3);
  b -= c; b -= a; b ^= (a<<10);
  c -= a; c -= b; c ^= (b>>15);

  return c;
}

__device__ float getNextRandom(const glm::ivec2 canvasSize, const uint32_t idx, uint32_t* consumedFloats, const float* floats, const uint32_t* scrambleConstants)
{
  uint32_t& consumed = consumedFloats[idx];

  const uint32_t totalFloats = ((canvasSize.x*canvasSize.y + RANDOM_DIMENSIONS-1) / RANDOM_DIMENSIONS)*PREGEN_RANDS*RANDOM_DIMENSIONS;
  const uint32_t scrambleConst = scrambleConstants[idx];

  const float f = floats[(PREGEN_RANDS*idx+consumed++) % totalFloats];

  const uint32_t i = mix(scrambleConst, idx, f * 4294967296);
  const float rf = 2.3283064365386963e-10f * i;

  return rf;
}

__device__ bool bboxIntersect(const AABB box, const float3 origin,
    const float3 inverseDirection, float& t)
{
  float3 tmin = make_float3(-BIGT, -BIGT, -BIGT), tmax = make_float3(BIGT, BIGT,
      BIGT);

  const float3 tdmin = (box.min - origin) * inverseDirection;
  const float3 tdmax = (box.max - origin) * inverseDirection;

  tmin = fminf(tdmin, tdmax);
  tmax = fmaxf(tdmin, tdmax);

  const float tmind = fmin_compf(tmin);
  const float tmaxd = fmin_compf(tmax);

  t = fminf(tmind, tmaxd);

  return tmaxd >= tmind && !(tmaxd < 0.f && tmind < 0.f);
}

__device__ bool rayTriangleIntersection(const Ray ray, const Triangle& triangle,
    float& t, float2& uv)
{
  /* Möller-Trumbore algorithm
   * https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
   */

  // TODO: Experiment with __ldg
  const float3 vertex0 = triangle.vertices[0].p;

  const float3 edge1 = triangle.vertices[1].p - vertex0;
  const float3 edge2 = triangle.vertices[2].p - vertex0;

  const float3 h = cross(ray.direction, edge2);
  const float a = dot(edge1, h);

  if (a > -INTERSECT_EPSILON && a < INTERSECT_EPSILON)
    return false;

  const float f = __fdividef(1.f, a);
  const float3 s = ray.origin - vertex0;
  const float u = f * dot(s, h);

  if (u < 0.f || u > 1.0f)
    return false;

  const float3 q = cross(s, edge1);
  const float v = f * dot(ray.direction, q);

  if (v < 0.0 || u + v > 1.0)
    return false;

  t = f * dot(edge2, q);

  if (t > INTERSECT_EPSILON)
  {
    uv = make_float2(u, v);
    return true;
  }
  else
    return false;
}

enum HitType
{
  ANY, CLOSEST
};

template<const HitType hitType>
__device__
RaycastResult rayCast(const Ray ray, const Node* bvh, const Triangle* triangles,
    const float maxT)
{
  float tMin = maxT;
  int32_t minTriIdx = -1;
  float2 minUV;
  RaycastResult result;
  const float3 inverseDirection = make_float3(1.f, 1.f, 1.f) / ray.direction;

  int32_t ptr = 0;
  unsigned int stack[16] { 0 };
  int32_t i = -1;
  float t = 0;
  float2 uv;
  bool getNextNode = true;

  while (ptr >= 0)
  {
    uint32_t currentNodeIdx = stack[ptr];
    Node currentNode = bvh[currentNodeIdx];

    if (currentNode.rightIndex == -1)
    {
      getNextNode = false;

      if (i >= currentNode.startTri
          && i < currentNode.startTri + currentNode.nTri)
      {
        if (rayTriangleIntersection(ray, triangles[i], t, uv))
        {

          if (t < tMin)
          {
            tMin = t;
            minTriIdx = i;
            minUV = uv;

            if (hitType == HitType::ANY)
              break;
          }
        }

        ++i;

        if (i >= currentNode.startTri + currentNode.nTri)
          getNextNode = true;

      }
      else
      {
        i = currentNode.startTri;
      }

    }
    else
    {
      const AABB leftBox = bvh[stack[ptr] + 1].bbox;
      const AABB rightBox = bvh[currentNode.rightIndex].bbox;

      float leftt, rightt;

      uint32_t hitMask =
          bboxIntersect(leftBox, ray.origin, inverseDirection, leftt) ?
              LEFT_HIT_BIT : 0x00;
      hitMask =
          bboxIntersect(rightBox, ray.origin, inverseDirection, rightt) ?
              hitMask | RIGHT_HIT_BIT : hitMask;

      // TODO: Push closer one last, don't intersect if closest hit is closer than box
      if ((hitMask & LEFT_HIT_BIT) != 0x00 && leftt < tMin)
      {
        stack[ptr] = currentNodeIdx + 1;
        ++ptr;
      }

      if ((hitMask & RIGHT_HIT_BIT) != 0x00 && rightt < tMin)
      {
        stack[ptr] = currentNode.rightIndex;
        ++ptr;
      }
    }

    if (getNextNode)
    {
      --ptr;
      i = -1;
    }

  }

  if (minTriIdx == -1)
    return result;

  result.point = ray.origin + ray.direction * tMin;
  result.t = tMin;
  result.triangleIdx = minTriIdx;
  result.uv = minUV;

  return result;
}

__device__ void writeToCanvas(const uint32_t x, const uint32_t y,
    const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize,
    const float3 data)
{
  const float4 out = make_float4(data.x, data.y, data.z, 1.f);
  surf2Dwrite(out, surfaceObj, (canvasSize.x - 1 - x) * sizeof(out), y);
  return;
}

__device__ float3 readFromCanvas(const uint32_t x, const uint32_t y,
    const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize)
{
  float4 in;
  surf2Dread(&in, surfaceObj, (canvasSize.x - 1 - x) * sizeof(in), y);
  const float3 ret = make_float3(in.x, in.y, in.z);

  return ret;
}

__global__ void logicKernel(const glm::ivec2 canvasSize, Queues queues,
    Paths paths, const Material* materials, const uint32_t* triangleMaterialIds)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const uint32_t idx = x + y * canvasSize.x;

  const float3 float3_zero = make_float3(0.f, 0.f, 0.f);

  if (x >= canvasSize.x  || y >= canvasSize.y)
    return;

  const RaycastResult result = paths.result[idx];
  const uint32_t rayNr = paths.rayNr[idx];

  if (!result || rayNr >= 5)
  {
    const uint32_t new_idx = atomicAdd(queues.newPathQueueSize, 1);
    queues.newPathQueue[new_idx] = idx;
    return;
  }else
  {
    const uint32_t new_idx = atomicAdd(queues.extensionQueueSize, 1);
    queues.extensionQueue[new_idx] = idx;
  }

  const Material material = materials[triangleMaterialIds[result.triangleIdx]];

  if (material.colorDiffuse != float3_zero)
  {
    const uint32_t new_idx = atomicAdd(queues.diffuseQueueSize, 1);
    queues.diffuseQueue[new_idx] = idx;
  }

  /*if (material.colorSpecular != float3_zero)
   {
   const uint32_t new_idx = atomicAdd(queues.specularQueueSize, 1);
   queues.specularQueue[new_idx] = idx;
   }*/

  return;
}

__global__ void writeToCanvas(const glm::ivec2 canvasSize,
    hipSurfaceObject_t canvas, Paths paths)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const int idx = x + y * canvasSize.x;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  const uint32_t currentPath = paths.pathNr[idx];
  const float3 newColor = paths.color[idx];
  const uint2 pixel = paths.pixel[idx];
  float3 oldColor = readFromCanvas(pixel.x, pixel.y, canvas, canvasSize);
  float3 blend = static_cast<float>(currentPath - 1) / currentPath * oldColor
      + 1.f / currentPath * newColor;

  writeToCanvas(pixel.x, pixel.y, canvas, canvasSize, blend);
}

typedef struct
{
  float3 col[3];
} float33;

__device__
inline void setZero(float33& m)
{
  m.col[0] = make_float3(0.0f, 0.0f, 0.0f);
  m.col[1] = make_float3(0.0f, 0.0f, 0.0f);
  m.col[2] = make_float3(0.0f, 0.0f, 0.0f);
}

__device__
inline float3 operator*(const float33 m, const float3 v)
{
  float3 res;
  res.x = m.col[0].x * v.x + m.col[1].x * v.y + m.col[2].x * v.z;
  res.y = m.col[0].y * v.x + m.col[1].y * v.y + m.col[2].y * v.z;
  res.z = m.col[0].z * v.x + m.col[1].z * v.y + m.col[2].z * v.z;

  return res;
}

__device__ float33 getBasis(const float3 n)
{

  float33 R;

  float3 Q = n;
  const float3 absq = abs(Q);
  float absqmin = fmin(absq);

  if (absq.x == absqmin)
    Q.x = 1;
  else if (absq.y == absqmin)
    Q.y = 1;
  else
    Q.z = 1;

  float3 T = normalize(cross(Q, n));
  float3 B = normalize(cross(n, T));

  R.col[0] = T;
  R.col[1] = B;
  R.col[2] = n;

  return R;
}

__global__ void diffuseKernel(const glm::ivec2 canvasSize, const Queues queues,
    Paths paths, const Triangle* triangles, const uint32_t* lightTriangleIds,
    const uint32_t lightTriangles, const uint32_t* triangleMaterialIds,
    const Material* materials, const Node* bvh)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const uint32_t idx = x + y * canvasSize.x;

  if (idx >= *queues.diffuseQueueSize)
    return;

  const float3 float3_zero = make_float3(0.f, 0.f, 0.f);
  const uint32_t pathIdx = queues.diffuseQueue[idx];

  const RaycastResult result = paths.result[pathIdx];
  const Material& material = materials[triangleMaterialIds[result.triangleIdx]];

  const Triangle triangle = triangles[result.triangleIdx];
  float3 hitNormal = triangle.normal();

  const float3 shadowRayOrigin = result.point + hitNormal * OFFSET_EPSILON;

  float3 brightness = make_float3(0.f, 0.f, 0.f);

  for (uint32_t i = 0; i < lightTriangles; ++i)
  {
    float pdf;
    float3 shadowPoint;

	const float r0 = getNextRandom(canvasSize, pathIdx, paths.randomNumbersConsumed, paths.randomFloats, paths.scrambleConstants);
	const float r1 = getNextRandom(canvasSize, pathIdx, paths.randomNumbersConsumed, paths.randomFloats, paths.scrambleConstants);

    triangles[lightTriangleIds[i]].sample(pdf, shadowPoint, r0, r1);

    const float3 shadowRayDirection = shadowPoint - shadowRayOrigin;
    const Ray shadowRay(shadowRayOrigin, normalize(shadowRayDirection));
    const float shadowRayLength = length(shadowRayDirection);

    const Triangle lightTriangle = triangles[lightTriangleIds[i]];
    const Material lightTriangleMaterial =
        materials[triangleMaterialIds[lightTriangleIds[i]]];
    const float3 lightEmission = lightTriangleMaterial.colorEmission;

    RaycastResult shadowResult = rayCast<HitType::ANY>(shadowRay, bvh, triangles, shadowRayLength);

    if ((shadowResult && shadowResult.t >= shadowRayLength + OFFSET_EPSILON)
        || !shadowResult)
    {
      const float cosOmega = __saturatef(
          dot(normalize(shadowRayDirection), hitNormal));
      const float cosL = __saturatef(
          dot(-normalize(shadowRayDirection), lightTriangle.normal()));

      brightness += 1.f / (shadowRayLength * shadowRayLength * pdf)
          * lightEmission * cosL * cosOmega;
    }
  }

  const float3 filteredAmbient = paths.throughput[pathIdx]
      * material.colorAmbient;
  const float3 filteredDiffuse = paths.throughput[pathIdx]
      * material.colorDiffuse;
  const float3 fiteredEmission = paths.throughput[pathIdx]
      * material.colorEmission;

  paths.color[pathIdx] += fiteredEmission + filteredAmbient
      + brightness / lightTriangles * filteredDiffuse / HIP_PI_F;
}

__global__ void newPathsKernel(const glm::ivec2 canvasSize, const Queues queues,
    Paths paths, const Camera camera)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const int idx = x + y * canvasSize.x;

  if (idx >= *queues.newPathQueueSize)
    return;

  const uint32_t pathIdx = queues.newPathQueue[idx];
  const uint2 pixel = paths.pixel[pathIdx];

  const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(pixel.x, pixel.y, canvasSize);
  const Ray ray = camera.generateRay(nic,
      static_cast<float>(canvasSize.x) / canvasSize.y);

  paths.ray[pathIdx] = ray;

  const uint32_t newExtensionIdx = atomicAdd(queues.extensionQueueSize, 1);
  queues.extensionQueue[newExtensionIdx] = pathIdx;
  paths.color[pathIdx] = make_float3(0.f, 0.f, 0.f);
  paths.throughput[pathIdx] = make_float3(1.f, 1.f, 1.f);
  paths.p[pathIdx] = 1.f;
  paths.rayNr[pathIdx] = 1;
  paths.pathNr[pathIdx] += 1;
}

inline __device__ float3 reflectionDirection(const float3 normal,
    const float3 incomingDirection)
{

  const float cosT = dot(incomingDirection, normal);

  return incomingDirection - 2 * cosT * normal;
}

__global__ void createExtensionKernel(const glm::ivec2 canvasSize,
    const Queues queues, Paths paths, const Triangle* triangles,
    const uint32_t* triangleMaterialIds, const Material* materials)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const uint32_t idx = x + y * canvasSize.x;

  if (idx >= *queues.extensionQueueSize)
    return;

  const uint32_t pathIdx = queues.extensionQueue[idx];

  const RaycastResult result = paths.result[pathIdx];
  const Triangle triangle = triangles[result.triangleIdx];
  const Material& material = materials[triangleMaterialIds[result.triangleIdx]];
  float3 hitNormal = triangle.normal();

  float33 B = getBasis(hitNormal);
  float3 extensionDir;

  do
  {
	const float r0 = getNextRandom(canvasSize, pathIdx, paths.randomNumbersConsumed, paths.randomFloats, paths.scrambleConstants);
	const float r1 = getNextRandom(canvasSize, pathIdx, paths.randomNumbersConsumed, paths.randomFloats, paths.scrambleConstants);

    extensionDir = make_float3(r0 * 2.0f - 1.0f, r1 * 2.0f - 1.0f, 0.f);
  } while ((extensionDir.x * extensionDir.x + extensionDir.y * extensionDir.y) >= 1);

  extensionDir.z = sqrt(1 - extensionDir.x * extensionDir.x - extensionDir.y * extensionDir.y);
  extensionDir = B * extensionDir;
  extensionDir = normalize(extensionDir); // Unnecessary
  const float3 extensionOrig = result.point + OFFSET_EPSILON * hitNormal;
  const Ray extensionRay(extensionOrig, extensionDir);

  float cosO = dot(extensionDir, hitNormal);
  float p = cosO * dot(extensionDir, hitNormal) * (1.f / HIP_PI_F);
  float3 throughput = material.colorDiffuse / HIP_PI_F* dot(extensionDir, hitNormal);

  paths.ray[pathIdx] = extensionRay;
  paths.throughput[pathIdx] = paths.throughput[pathIdx] * throughput;
  paths.p[pathIdx] *= p;
  paths.rayNr[pathIdx] += 1;
}

/*__global__ void
 specularKernel(
 const glm::ivec2 canvasSize,
 const Queues queues,
 Paths paths,
 const Triangle* triangles,
 const uint32_t* lightTriangleIds,
 const uint32_t  lightTriangles,
 const uint32_t* triangleMaterialIds,
 const Material* materials,
 const Node* bvh)
 {
 const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
 const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
 const uint32_t idx = x + y * canvasSize.x;

 if (idx >= *queues.specularQueueSize - 1)
	 return;

 const float3 float3_zero = make_float3(0.f, 0.f, 0.f);
 const uint32_t pathIdx = queues.specularQueue[idx];

 const Ray hitRay = paths.ray[pathIdx];
 const RaycastResult hitResult = paths.result[pathIdx];

 const Triangle triangle = triangles[hitResult.triangleIdx];
 float3 hitNormal = triangle.normal();

 const float3 reflectionRayOrigin = hitResult.point + hitNormal * OFFSET_EPSILON;
 const float3 reflectionRayDir = reflectionDirection(hitNormal, hitRay.direction);

 const Ray reflectionRay(reflectionRayOrigin, reflectionRayDir);

 const uint32_t newRayIdx = atomicAdd(paths.secondaryPathCount, 1);
 paths.secondaryRays[newRayIdx] = reflectionRay;
 paths.secondaryPixels[newRayIdx] = currentPixel;
 paths.secondaryFilters[newRayIdx] = paths.primaryFilters[primaryIdx];
}*/

__global__ void resetAllPaths(Paths paths, Camera camera,
    const glm::fvec2 canvasSize)
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  const int idx = x + y * canvasSize.x;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(x,
      y, canvasSize);
  const Ray ray = camera.generateRay(nic,
      static_cast<float>(canvasSize.x) / canvasSize.y);

  paths.ray[idx] = ray;
  paths.color[idx] = make_float3(0.f, 0.f, 0.f);
  paths.throughput[idx] = make_float3(1.f, 1.f, 1.f);
  paths.pixel[idx] = make_uint2(x, y);
  paths.p[idx] = 1.f;
  paths.rayNr[idx] = 1;
  paths.pathNr[idx] = 1;
  //paths.randomNumbersConsumed[idx] = 0;
}

__global__ void castRays(Paths paths, const glm::ivec2 canvasSize,
    const Triangle* triangles, const Node* bvh, const Material* materials,
    const unsigned int* traingelMaterialIds)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int idx = x + y * canvasSize.x;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  const Ray ray = paths.ray[idx];
  RaycastResult result = rayCast<HitType::CLOSEST>(ray, bvh, triangles, BIGT);
  paths.result[idx] = result;
}

__global__ void
testRnd(
    const hipSurfaceObject_t canvas,
    const glm::ivec2 canvasSize,
    uint32_t* randomsConsumed,
    float* floats,
    uint32_t* scrambleConstants
    )
{
  const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

  const uint32_t idx = x + y*canvasSize.x;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  const float f0 = getNextRandom(canvasSize, idx, randomsConsumed, floats, scrambleConstants);
  const float f1 = getNextRandom(canvasSize, idx, randomsConsumed, floats, scrambleConstants);

  if (f0 < 0.5f)
	  return;

  writeToCanvas(x, y, canvas, canvasSize, make_float3(f0, f1, 0.f));

  return;
}

void CudaRenderer::reset()
{
  queues.reset();

  dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  dim3 grid((lastSize.x + block.x - 1) / block.x, (lastSize.y + block.y - 1) / block.y);

  resetAllPaths<<<grid, block>>>(paths, lastCamera, lastSize);
  CUDA_CHECK(hipDeviceSynchronize());
}

void CudaRenderer::resize(const glm::ivec2 size)
{
  queues.resize(size);
  paths.resize(size);

  lastSize = size;
  callcntr = 0;

  CURAND_CHECK(hiprandSetQuasiRandomGeneratorDimensions(rndGen, RANDOM_DIMENSIONS));

  dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  dim3 grid((size.x + block.x - 1) / block.x, (size.y + block.y - 1) / block.y);

  uint32_t* hostScrambleConstants32;

  CUDA_CHECK(hiprandGetScrambleConstants32(&hostScrambleConstants32));
  CUDA_CHECK(hipMemcpy(paths.scrambleConstants, hostScrambleConstants32, size.x*size.y, hipMemcpyHostToDevice));

  for (int offset = 0; offset < PREGEN_RANDS*size.x*size.y; offset += PREGEN_RANDS*RANDOM_DIMENSIONS)
	  CURAND_CHECK(hiprandGenerateUniform(rndGen, paths.randomFloats+offset, PREGEN_RANDS*RANDOM_DIMENSIONS));

  CUDA_CHECK(hipMemset(paths.randomNumbersConsumed, 0, size.x*size.y*sizeof(uint32_t)));

  CUDA_CHECK(hipDeviceSynchronize());

  reset();
}

CudaRenderer::CudaRenderer()
    : lastCamera(), lastSize(), callcntr(0)
{
  uint32_t cudaDeviceCount = 0;
  int cudaDevices[8];
  uint32_t cudaDevicesCount = 8;

  hipGLGetDevices(&cudaDeviceCount, cudaDevices, cudaDevicesCount,
      hipGLDeviceListCurrentFrame);

  if (cudaDeviceCount < 1)
  {
    std::cout << "No CUDA devices found" << std::endl;
    throw std::runtime_error("No CUDA devices available");
  }

  CUDA_CHECK(hipSetDevice(cudaDevices[0]));

  CURAND_CHECK(hiprandCreateGenerator(&rndGen, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32));

  resize(glm::ivec2(WWIDTH, WHEIGHT));
}

CudaRenderer::~CudaRenderer()
{
  queues.release();
  paths.release();
  CURAND_CHECK(hiprandDestroyGenerator(rndGen));
  CUDA_CHECK(hipDeviceReset());
}

void CudaRenderer::pathTraceToCanvas(GLTexture& canvas, const Camera& camera,
    Model& model)
{
  if (model.getNTriangles() == 0)
    return;

  const glm::ivec2 canvasSize = canvas.getSize();
  const bool diffCamera = std::memcmp(&camera, &lastCamera, sizeof(Camera));
  const bool diffSize = (canvasSize != lastSize);
  const auto surfaceObj = canvas.getCudaMappedSurfaceObject();

  const dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  const dim3 grid((canvasSize.x + block.x - 1) / block.x, (canvasSize.y + block.y - 1) / block.y);

  if (diffCamera != 0 || diffSize != 0)
  {
    lastCamera = camera;

    reset();
  }

  /*testRnd<<<grid, block>>>(surfaceObj,
		    canvasSize,
		    paths.randomNumbersConsumed,
		    paths.randomFloats,
		    paths.scrambleConstants);*/


  castRays<<<grid, block>>>(paths, canvasSize, model.getDeviceTriangles(),
      model.getDeviceBVH(), model.getDeviceMaterials(),
      model.getDeviceTriangleMaterialIds());

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemset(queues.extensionQueueSize, 0, sizeof(uint32_t)));

  logicKernel<<<grid, block>>>(canvasSize, queues, paths,
      model.getDeviceMaterials(), model.getDeviceTriangleMaterialIds());

  CUDA_CHECK(hipDeviceSynchronize());

  diffuseKernel<<<grid, block>>>(canvasSize, queues, paths,
      model.getDeviceTriangles(), model.getDeviceLightIds(), model.getNLights(),
      model.getDeviceTriangleMaterialIds(), model.getDeviceMaterials(),
      model.getDeviceBVH());

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemset(queues.diffuseQueueSize, 0, sizeof(uint32_t)));

  /*specularKernel<<<grid, block>>>(
	canvasSize,
	queues,
	paths,
	model.getDeviceTriangles(),
	model.getDeviceLightIds(),
	model.getNLights(),
	model.getDeviceTriangleMaterialIds(),
	model.getDeviceMaterials(),
	model.getDeviceBVH()
	);

  CUDA_CHECK(hipDeviceSynchronize());
  *queues.specularQueueSize = 0;*/

  writeToCanvas<<<grid, block>>>(canvasSize, surfaceObj, paths);

  CUDA_CHECK(hipDeviceSynchronize());

  createExtensionKernel<<<grid, block>>>(canvasSize, queues, paths,
      model.getDeviceTriangles(), model.getDeviceTriangleMaterialIds(),
      model.getDeviceMaterials());

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemset(queues.extensionQueueSize, 0, sizeof(uint32_t)));

  newPathsKernel<<<grid, block>>>(canvasSize, queues, paths, camera);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemset(queues.newPathQueueSize, 0, sizeof(uint32_t)));

  canvas.cudaUnmap();
}

